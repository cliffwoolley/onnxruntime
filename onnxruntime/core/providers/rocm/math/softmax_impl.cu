#include "hip/hip_runtime.h"
/**
 * Copyright (c) 2016-present, Facebook, Inc.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* Modifications Copyright (c) Microsoft. */

// The code below is mostly copied from Pytorch PersistentSoftmax.cuh
#include "hip/hip_runtime.h"

#include "core/providers/rocm/cu_inc/common.cuh"
#include "core/providers/rocm/math/softmax_warpwise_impl.cuh"
#include "core/providers/rocm/math/softmax_blockwise_impl.cuh"
#include "core/providers/rocm/math/softmax.h"

#include <limits>

namespace onnxruntime {
namespace rocm {

template <typename input_t, typename output_t, typename acc_t, bool is_log_softmax>
void dispatch_warpwise_softmax_forward(hipStream_t stream, output_t* dst, const input_t* src, int softmax_elements, int softmax_elements_stride, int batch_count) {
  if (softmax_elements == 0) {
    return;
  } else {
    int log2_elements = log2_ceil(softmax_elements);
    const int next_power_of_two = 1 << log2_elements;

    // This value must match the WARP_SIZE constexpr value computed inside softmax_warp_forward.
    int warp_size = (next_power_of_two < GPU_WARP_SIZE_HOST) ? next_power_of_two : GPU_WARP_SIZE_HOST;

    // This value must match the WARP_BATCH constexpr value computed inside softmax_warp_forward.
    int batches_per_warp = 1;
    // use 256 threads per block to maximimize gpu utilization
    constexpr int threads_per_block = 256;

    int warps_per_block = (threads_per_block / warp_size);
    int batches_per_block = warps_per_block * batches_per_warp;
    int blocks = (batch_count + batches_per_block - 1) / batches_per_block;
    dim3 threads(warp_size, warps_per_block, 1);
    // Launch code would be more elegant if C++ supported FOR CONSTEXPR
    switch (log2_elements) {
      #define LAUNCH_SOFTMAX_WARP_FORWARD(L2E)                                                         \
        case L2E:                                                                                      \
          softmax_warp_forward<input_t, output_t, acc_t, L2E, is_log_softmax>                          \
              <<<dim3(blocks), dim3(threads), 0, stream>>>(dst, src, batch_count,                      \
                                                          softmax_elements_stride, softmax_elements);  \
          break;
      LAUNCH_SOFTMAX_WARP_FORWARD(0);   // 1
      LAUNCH_SOFTMAX_WARP_FORWARD(1);   // 2
      LAUNCH_SOFTMAX_WARP_FORWARD(2);   // 4
      LAUNCH_SOFTMAX_WARP_FORWARD(3);   // 8
      LAUNCH_SOFTMAX_WARP_FORWARD(4);   // 16
      LAUNCH_SOFTMAX_WARP_FORWARD(5);   // 32
      LAUNCH_SOFTMAX_WARP_FORWARD(6);   // 64
      LAUNCH_SOFTMAX_WARP_FORWARD(7);   // 128
      LAUNCH_SOFTMAX_WARP_FORWARD(8);   // 256
      LAUNCH_SOFTMAX_WARP_FORWARD(9);   // 512
      LAUNCH_SOFTMAX_WARP_FORWARD(10);  // 1024
      default:
        break;
    }
  }
}

template <typename input_t, typename output_t, typename acc_t, bool is_log_softmax>
void dispatch_blockwise_softmax_forward(hipStream_t stream, output_t* output, const input_t* input, int softmax_elements,
                                        int input_stride, int output_stride, int batch_count) {
  dim3 grid(batch_count);
  constexpr int ILP = sizeof(float4) / sizeof(input_t);
  dim3 block = SoftMax_getBlockSize(ILP, softmax_elements);
  if (is_log_softmax) {
    softmax_block_forward<ILP, input_t, acc_t, output_t, LogSoftMaxForwardEpilogue>
        <<<grid, block, block.x * sizeof(acc_t), stream>>>(output, const_cast<input_t*>(input),
                                                           softmax_elements, input_stride, output_stride);
  } else {
    softmax_block_forward<ILP, input_t, acc_t, output_t, SoftMaxForwardEpilogue>
        <<<grid, block, block.x * sizeof(acc_t), stream>>>(output, const_cast<input_t*>(input),
                                                           softmax_elements, input_stride, output_stride);
  }
}

#define SPECIALIZED_SOFTMAX_IMPL(input_t, output_t, acc_t)                             \
  template void dispatch_warpwise_softmax_forward<input_t, output_t, acc_t, false>(    \
      hipStream_t stream, output_t * dst, const input_t* src, int softmax_elements,    \
      int softmax_elements_stride, int batch_count);                                   \
  template void dispatch_warpwise_softmax_forward<input_t, output_t, acc_t, true>(     \
      hipStream_t stream, output_t * dst, const input_t* src, int softmax_elements,    \
      int softmax_elements_stride, int batch_count);                                   \
  template void dispatch_blockwise_softmax_forward<input_t, output_t, acc_t, false>(   \
      hipStream_t stream, output_t * output, const input_t* src, int softmax_elements, \
      int input_stride, int output_stride, int batch_count);                           \
  template void dispatch_blockwise_softmax_forward<input_t, output_t, acc_t, true>(    \
      hipStream_t stream, output_t * output, const input_t* src, int softmax_elements, \
      int input_stride, int output_stride, int batch_count);

SPECIALIZED_SOFTMAX_IMPL(float, float, float)
SPECIALIZED_SOFTMAX_IMPL(half, half, float)
SPECIALIZED_SOFTMAX_IMPL(double, double, double)
SPECIALIZED_SOFTMAX_IMPL(BFloat16, BFloat16, float)

}  // namespace rocm
}  // namespace onnxruntime
